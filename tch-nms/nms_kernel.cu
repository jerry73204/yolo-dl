#include "hip/hip_runtime.h"
/* Copyright (c) 2018, Grégoire Payen de La Garanderie, Durham University
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <ATen/ATen.h>
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// Hard-coded maximum. Increase if needed.
#define MAX_COL_BLOCKS 1000

#define DIVUP(m, n) (((m) + (n)-1) / (n))
int64_t const threadsPerBlock = sizeof(unsigned long long) * 8;

// The functions below originates from Fast R-CNN
// See https://github.com/rbgirshick/py-faster-rcnn
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License
// Written by Shaoqing Ren

template <typename scalar_t>
__device__ inline scalar_t devIoU(scalar_t const *const a,
                                  scalar_t const *const b) {
  scalar_t left = max(a[0], b[0]), right = min(a[2], b[2]);
  scalar_t top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  scalar_t width = max(right - left, 0.f), height = max(bottom - top, 0.f);
  scalar_t interS = width * height;
  scalar_t Sa = (a[2] - a[0]) * (a[3] - a[1]);
  scalar_t Sb = (b[2] - b[0]) * (b[3] - b[1]);
  return interS / (Sa + Sb - interS);
}

template <typename scalar_t>
__global__ void
nms_kernel(const int64_t n_boxes, const scalar_t nms_overlap_thresh,
           const scalar_t *dev_boxes, const int64_t *idx, int64_t *dev_mask) {
  const int64_t row_start = blockIdx.y;
  const int64_t col_start = blockIdx.x;

  const int row_size =
      min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
      min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ scalar_t block_boxes[threadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        dev_boxes[idx[(threadsPerBlock * col_start + threadIdx.x)] * 4 + 3];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const scalar_t *cur_box = dev_boxes + idx[cur_box_idx] * 4;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 4) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

__global__ void nms_collect(const int64_t boxes_num, const int64_t col_blocks,
                            int64_t top_k, const int64_t *idx,
                            const int64_t *mask, int64_t *keep,
                            int64_t *parent_object_index,
                            int64_t *num_to_keep) {
  int64_t remv[MAX_COL_BLOCKS];
  int64_t num_to_keep_ = 0;

  for (int i = 0; i < col_blocks; i++) {
    remv[i] = 0;
  }

  for (int i = 0; i < boxes_num; ++i) {
    parent_object_index[i] = 0;
  }

  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      int64_t idxi = idx[i];
      keep[num_to_keep_] = idxi;
      const int64_t *p = &mask[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
      for (int j = i; j < boxes_num; j++) {
        int nblockj = j / threadsPerBlock;
        int inblockj = j % threadsPerBlock;
        if (p[nblockj] & (1ULL << inblockj))
          parent_object_index[idx[j]] = num_to_keep_ + 1;
      }
      parent_object_index[idx[i]] = num_to_keep_ + 1;

      num_to_keep_++;

      if (num_to_keep_ == top_k)
        break;
    }
  }

  // Initialize the rest of the keep array to avoid uninitialized values.
  for (int i = num_to_keep_; i < boxes_num; ++i)
    keep[i] = 0;

  *num_to_keep = min(top_k, num_to_keep_);
}

#define CHECK_CONTIGUOUS(x)                                                    \
  AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")

extern "C" {
void nms_cuda_forward_ffi(at::Tensor **keep, at::Tensor **num_to_keep,
                          at::Tensor **parent_object_index, at::Tensor *boxes,
                          at::Tensor *idx, float nms_overlap_thresh,
                          unsigned long top_k) {
  const auto boxes_num = boxes->size(0);
  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  AT_ASSERTM(col_blocks < MAX_COL_BLOCKS,
             "The number of column blocks must be less than MAX_COL_BLOCKS. "
             "Increase the MAX_COL_BLOCKS constant if needed.");

  auto longOptions =
      torch::TensorOptions().device(torch::kCUDA).dtype(torch::kLong);
  auto mask = at::empty({boxes_num * col_blocks}, longOptions);

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);

  AT_ASSERTM(boxes->is_contiguous(), "boxes must be contiguous");
  AT_ASSERTM(idx->is_contiguous(), "idx must be contiguous");
  AT_ASSERTM(mask.is_contiguous(), "mask must be contiguous");

  AT_DISPATCH_FLOATING_TYPES(boxes->type(), "nms_cuda_forward", ([&] {
                               nms_kernel<<<blocks, threads>>>(
                                   boxes_num, (scalar_t)nms_overlap_thresh,
                                   boxes->data<scalar_t>(),
                                   idx->data<int64_t>(), mask.data<int64_t>());
                             }));

  at::Tensor keep_ = at::empty({boxes_num}, longOptions);
  at::Tensor num_to_keep_ = at::empty({}, longOptions);
  at::Tensor parent_object_index_ = at::empty({boxes_num}, longOptions);

  nms_collect<<<1, 1>>>(boxes_num, col_blocks, top_k, idx->data<int64_t>(),
                        mask.data<int64_t>(), keep_.data<int64_t>(),
                        parent_object_index_.data<int64_t>(),
                        num_to_keep_.data<int64_t>());

  *keep = new at::Tensor(keep_);
  *num_to_keep = new at::Tensor(num_to_keep_);
  *parent_object_index = new at::Tensor(parent_object_index_);
}
}